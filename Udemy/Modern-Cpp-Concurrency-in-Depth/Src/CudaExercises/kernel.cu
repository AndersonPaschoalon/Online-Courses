#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdio.h>
#include <stdlib.h>
#include <cstring>
#include <time.h>

// Section 88

__global__ void hello_cuda()
{
	printf("Hello CUDA!\n");
}

void section88()
{
	int nx = 16;
	int ny = 4;

	dim3 block(8, 2, 1);
	dim3 grid(nx/block.x, ny/block.y, 1);


	hello_cuda << <grid, block >> > ();
	hipDeviceSynchronize();
	hipDeviceReset();
}

// Section 89

__global__ void print_threadIds()
{
	printf("threadIdx.x:%d, threadIdx.y:%d, threadIdx.z:%d", 
		threadIdx.x, threadIdx.y, threadIdx.z);

}

// Section 90

__global__ void print_details()
{
	printf("-- blockDim.x:%d, blockDim.y:%d, blockDim.z:%d",
		blockDim.x, blockDim.y, blockDim.z);
	printf("-- gridDim.x:%d, gridDim.y:%d, gridDim.z:%d",
		gridDim.x, gridDim.y, gridDim.z);
}

void section89()
{
	int nx = 16;
	int ny = 16;
	dim3 block(8, 8);
	dim3 grid(nx/block.x, ny/block.y);
	print_threadIds << <grid, block >> > ();

	hipDeviceSynchronize();
	hipDeviceReset();
}

void section90()
{
	int nx = 16;
	int ny = 16;
	dim3 block(8, 8);
	dim3 grid(nx / block.x, ny / block.y);
	print_threadIds << <grid, block >> > ();

	hipDeviceSynchronize();
	hipDeviceReset();
}

//  Section 91

__global__ void unique_idx_calc_threadIdx(int* input)
{
	int tid = threadIdx.x;
	printf("threadIdx.x:%d, value:%d\n", tid, input[tid]);
}
__global__ void unique_gid_calculation(int* input)
{
	int tid = threadIdx.x;
	int offset = blockIdx.x * blockDim.x;
	int gid = tid + offset;
	printf("blockIdx.x:%d, threadIdx.x:%d, gid:%d, value:%d\n", 
		blockIdx.x, threadIdx.x, gid, input[gid]);
}


// Section 91

int section91_a()
{
	int array_size = 8;
	int array_byte_size = sizeof(int)*array_size;
	int h_data[] = {23, 9, 4, 53, 65, 12, 1, 33};

	for (int i = 0; i < array_size; i++)
	{
		printf("%d ", h_data[i]);
	}
	printf("\n \n ");

	int* d_data;
	hipMalloc((void**)&d_data, array_byte_size);
	hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);

	dim3 block(8);
	dim3 grid(1);

	unique_idx_calc_threadIdx << <grid, block >> > (d_data);
	hipDeviceSynchronize();
	hipDeviceReset();
	return 0;
}

int section91_b()
{
	int array_size = 8;
	int array_byte_size = sizeof(int) * array_size;
	int h_data[] = { 23, 9, 4, 53, 65, 12, 1, 33 };

	for (int i = 0; i < array_size; i++)
	{
		printf("%d ", h_data[i]);
	}
	printf("\n \n ");

	int* d_data;
	hipMalloc((void**)&d_data, array_byte_size);
	hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);

	dim3 block(4);
	dim3 grid(2);

	unique_idx_calc_threadIdx << <grid, block >> > (d_data);
	hipDeviceSynchronize();
	hipDeviceReset();
	return 0;
}

int section91_c()
{
	int array_size = 16;
	int array_byte_size = sizeof(int) * array_size;
	int h_data[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16};

	for (int i = 0; i < array_size; i++)
	{
		printf("%d ", h_data[i]);
	}
	printf("\n \n ");

	int* d_data;
	hipMalloc((void**)&d_data, array_byte_size);
	hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);

	dim3 block(4);
	dim3 grid(4);

	unique_gid_calculation << <grid, block >> > (d_data);
	hipDeviceSynchronize();
	hipDeviceReset();
	return 0;
}

// Section 92

__global__ void unique_gid_calculation_2d_a(int* input)
{
	int tid = threadIdx.x;
	int offset = blockIdx.x * blockDim.x;
	int gid = tid + offset;
	printf("blockIdx.x:%d, blockIdx.y:%d, threadIdx.x:%d, gid:%d, value:%d\n",
		blockIdx.x, blockIdx.y, tid, gid, input[gid]);
}

__global__ void unique_gid_calculation_2d_b(int* data)
{
	int tid = threadIdx.x;
	int block_offset = blockIdx.x * blockDim.x;
	int row_offset = blockDim.x * gridDim.x * blockIdx.y;
	int gid = row_offset + block_offset + tid;

	printf("blockIdx.x:%d, blockIdx.y:%d, threadIdx.x:%d, gid:%d, value:%d\n",
		blockIdx.x, blockIdx.y, tid, gid, data[gid]);
}

int section92_a()
{
	int array_size = 16;
	int array_byte_size = sizeof(int) * array_size;
	int h_data[] = { 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16 };

	for (int i = 0; i < array_size; i++)
	{
		printf("%d ", h_data[i]);
	}
	printf("\n \n ");

	int* d_data;
	hipMalloc((void**)&d_data, array_byte_size);
	hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);

	dim3 block(4);
	dim3 grid(2, 2);

	unique_gid_calculation_2d_a << <grid, block >> > (d_data);
	hipDeviceSynchronize();
	hipDeviceReset();
	return 0;
}

int section92_b()
{
	int array_size = 16;
	int array_byte_size = sizeof(int) * array_size;
	int h_data[] = { 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16 };

	for (int i = 0; i < array_size; i++)
	{
		printf("%d ", h_data[i]);
	}
	printf("\n \n ");

	int* d_data;
	hipMalloc((void**)&d_data, array_byte_size);
	hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);

	dim3 block(4);
	dim3 grid(2, 2);

	unique_gid_calculation_2d_b << <grid, block >> > (d_data);
	hipDeviceSynchronize();
	hipDeviceReset();
	return 0;
}

// Section 93

__global__ void unique_gid_calculation_2d_2d(int* data)
{
	int tid = blockDim.x * threadIdx.y + threadIdx.x;

	int num_threas_in_a_block = blockDim.x * blockDim.y;
	int block_offset = blockIdx.x * num_threas_in_a_block;

	int num_threads_in_a_row = num_threas_in_a_block * gridDim.x;
	int row_offset = num_threads_in_a_row * blockIdx.y;

	int gid = tid + block_offset + row_offset;


	printf("blockIdx.x:%d, blockIdx.y:%d, threadIdx.x:%d, gid:%d, value:%d\n",
		blockIdx.x, blockIdx.y, tid, gid, data[gid]);
}

int section93()
{
	int array_size = 16;
	int array_byte_size = sizeof(int) * array_size;
	int h_data[] = { 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16 };

	for (int i = 0; i < array_size; i++)
	{
		printf("%d ", h_data[i]);
	}
	printf("\n \n ");

	int* d_data;
	hipMalloc((void**)&d_data, array_byte_size);
	hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);

	dim3 block(2, 2);
	dim3 grid(2, 2);

	unique_gid_calculation_2d_2d << <grid, block >> > (d_data);
	hipDeviceSynchronize();
	hipDeviceReset();
	return 0;
}


// 95

__global__ void mem_trs_test(int* input)
{
	int gid = blockIdx.x * blockDim.x + threadIdx.x;
	printf("tid : %d, gid:%d, value:%d, \n", threadIdx.x, gid, input[gid]);
}

int section95()
{
	int size = 128;
	int byte_size = size * sizeof(int);
	int* h_input;
	h_input = (int*)malloc(byte_size);

	time_t t;
	srand((unsigned)time(&t));
	for (int i = 0; i < size; i++)
	{
		h_input[i] = (int)(rand() & 0xff);
	}
	int* d_input;
	hipMalloc((void**)&d_input, byte_size);
	hipMemcpy(d_input, h_input, byte_size, hipMemcpyHostToDevice);

	dim3 block(64);
	dim3 grid(2);

	mem_trs_test << <grid, block >> > (d_input);
	hipDeviceSynchronize();

	hipFree(d_input);
	free(h_input);


	hipDeviceReset();
	return 0;
}

int main()
{
	bool sec88 = false;
	bool sec89 = false;
	bool sec90 = false;
	bool sec91_a = false;
	bool sec91_b = false;
	bool sec91_c = false;
	bool sec92_a = false;
	bool sec92_b = false;
	bool sec93 = true;
	bool sec95 = true;

	if (sec88) section88();
	if (sec89) section89();
	if (sec90) section90();
	if (sec91_a) section91_a();
	if (sec91_b) section91_b();
	if (sec91_c) section91_c();
	if (sec92_a) section92_a();
	if (sec92_b) section92_b();
	if (sec93) section93();
	if (sec95) section95();

	for (int i = 0; i < 1000; i++)
	{
		section93();
	}

	return 0;
}
